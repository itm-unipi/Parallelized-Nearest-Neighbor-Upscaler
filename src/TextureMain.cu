#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#define STB_IMAGE_IMPLEMENTATION
#include "../lib/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../lib/stb_image_write.h"

#include "Headers.h"

using namespace std;

void saveImageToPNG(const char* filename, const float* imageData, int width, int height) 
{
    // create the uint8_t array
    uint8_t* byteImageData = new uint8_t[width * height * 4];

    // convert from float to uint8_t ([0, 1] -> [0, 255])
    for (int i = 0; i < width * height * 4; ++i)
        byteImageData[i] = (uint8_t)(imageData[i] * 255);

    // save the image
    stbi_write_png(filename, width, height, 4, byteImageData, width * 4);
    delete[] byteImageData;
}

__global__ void copyImage(hipTextureObject_t texObj, float* copiedImage, uint32_t width, uint32_t height, uint8_t bytePerPixel, size_t originalSize)
{
    uint32_t oldIndex = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint32_t newIndex = ((blockIdx.x * blockDim.x) + threadIdx.x) * bytePerPixel;

    if (newIndex < originalSize) {
        // convert the position in a matrix notation
        uint32_t x = oldIndex / width;
        uint32_t y = oldIndex - (x * width);

        // convert to normalized coordinates
        float u = y / (float)width;
        float v = x / (float)height;

        // copy the pixel
        float4 pixelToCopy = tex2D<float4>(texObj, u, v);
        copiedImage[newIndex] = pixelToCopy.x;
        copiedImage[newIndex + 1] = pixelToCopy.y;
        copiedImage[newIndex + 2] = pixelToCopy.z;
        copiedImage[newIndex + 3] = 1.0f;
    }
}

int main(int argc, char* argv[]) 
{
    int channel = Channels::RGB_ALPHA;
    string inputImageName;
    uint8_t upscaleFactor;

    // read image name from input parameters
    if (argc >= 2) {
        inputImageName = argv[1];
        upscaleFactor = atoi(argv[2]);
    }
    else {
        inputImageName = "img/in-small.png";
        upscaleFactor = 2;
    }

    // open the image
    uint32_t width, height, bytePerPixel;
    stbi_ldr_to_hdr_gamma(1.0f);
    float* data = stbi_loadf(inputImageName.c_str(), (int*)&width, (int*)&height, (int*)&bytePerPixel, channel);

    if (!data) {
        cout << "[-] Image not found" << endl;
        return -1;
    }

    // ----------------------------------- Texture Setup -----------------------------------------

    // allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);

    // copy data located at address h_data in host memory to device memory
    const size_t spitch = width * 4 * sizeof(float);
    hipMemcpy2DToArray(cuArray, 0, 0, data, spitch, width * 4 * sizeof(float), height, hipMemcpyHostToDevice);

    // specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    // create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    // -------------------------------------------------------------------------------------------

    // create array for the copied image
    size_t originalSize = height * width * bytePerPixel;
    float* copiedImage = new float[originalSize];

    // allocate GPU memory for output array 
    float* d_out;
    hipMalloc((void**)&d_out, originalSize * sizeof(float));

    // define resources for the execution
    dim3 block(128, 1, 1);
    dim3 grid(((width * height) + 127) / 128, 1, 1);

    // run the kernel
    copyImage << <grid, block >> > (texObj, d_out, width, height, bytePerPixel, originalSize);

    // retrieve result
    hipDeviceSynchronize();
    hipMemcpy(copiedImage, d_out, originalSize * sizeof(float), hipMemcpyDeviceToHost);

    // save image
    saveImageToPNG("img/TEST.png", copiedImage, width, height);

    // free device memory
    hipDestroyTextureObject(texObj);
    hipFreeArray(cuArray);
    hipFree(d_out);
    delete[] copiedImage;

    // Free host memory
    stbi_image_free(data);
    return 0;
}
