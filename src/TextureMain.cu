#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#define STB_IMAGE_IMPLEMENTATION
#include "../lib/stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "../lib/stb_image_write.h"

#include "Headers.h"
#include "GpuTimer.cuh"

using namespace std;

__global__ void upscaleImage(hipTextureObject_t texObj, uint8_t* upscaledImage, uint32_t width, uint32_t height, uint8_t bytePerPixel, size_t originalSize, uint32_t upscaleFactor)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < height && j < width) {
        // compute the position of the first pixel to duplicate in upscaled image
        uint32_t newi = i * upscaleFactor;
        uint32_t newj = j * upscaleFactor;
        uint32_t upscaledWidth = width * upscaleFactor;

        // convert to normalized coordinates
        float u = j / (float)width;
        float v = i / (float)height;

        // iterate the pixel to duplicate in upscaled image
        for (int m = newi; m < newi + upscaleFactor; m++) {
            for (int n = newj; n < newj + upscaleFactor; n++) {
                // compute the pixel position in the upscaled image vector
                uint32_t newIndex = (m * upscaledWidth + n) * bytePerPixel;

                // copy the pixel
                uchar4  pixelToCopy = tex2D<uchar4>(texObj, u, v);
                memcpy(&upscaledImage[newIndex], &pixelToCopy, sizeof(uchar4));
            }
        }
    }
}

int main(int argc, char* argv[]) 
{
    int channel = Channels::RGB_ALPHA;
    string inputImageName;
    uint8_t upscaleFactor;

    // read image name from input parameters
    if (argc >= 2) {
        inputImageName = argv[1];
        upscaleFactor = atoi(argv[2]);
    }
    else {
        inputImageName = "img/in-small.png";
        upscaleFactor = 2;
    }

    // open the image
    uint32_t width, height, bytePerPixel;
    uint8_t* data = stbi_load(inputImageName.c_str(), (int*)&width, (int*)&height, (int*)&bytePerPixel, channel);

    if (!data) {
        cout << "[-] Image not found" << endl;
        return -1;
    }

    // event based timer
    GpuTimer timer;

    // ----------------------------------- Texture Setup -----------------------------------------

    // allocate CUDA array in device memory
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray_t cuArray;
    hipMallocArray(&cuArray, &channelDesc, width, height);

    // copy data located at address data in host memory to device memory
    const size_t spitch = width * bytePerPixel * sizeof(uint8_t);
    hipMemcpy2DToArray(cuArray, 0, 0, data, spitch, width * bytePerPixel * sizeof(uint8_t), height, hipMemcpyHostToDevice);

    // specify texture
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    // specify texture object parameters
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeWrap;
    texDesc.addressMode[1] = hipAddressModeWrap;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1;

    // create texture object
    hipTextureObject_t texObj = 0;
    hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

    // -------------------------------------------------------------------------------------------

    // create array for the copied image
    size_t originalSize = height * width * bytePerPixel;
    size_t upscaledSize = originalSize * upscaleFactor * upscaleFactor;
    uint8_t* copiedImage = new uint8_t[upscaledSize];

    // allocate GPU memory for output array 
    uint8_t* d_out;
    hipMalloc((void**)&d_out, upscaledSize);

    // define resources for the execution
    dim3 block(32, 16);
    dim3 grid((height + block.x - 1) / block.x, (width + block.y - 1) / block.y);

    // run the kernel
    timer.start();
    upscaleImage << <grid, block >> > (texObj, d_out, width, height, bytePerPixel, originalSize, upscaleFactor);
    timer.stop();
    cout << "[+] (GPU) Time needed: " << timer.getElapsedMilliseconds() << "ms" << endl;

    // retrieve result
    hipDeviceSynchronize();
    hipMemcpy(copiedImage, d_out, upscaledSize, hipMemcpyDeviceToHost);

    // save image
    if (stbi_write_png("img/TEST.png", width * upscaleFactor, height * upscaleFactor, bytePerPixel, copiedImage, width * upscaleFactor * bytePerPixel))
        cout << "[+] Image saved successfully" << endl;

    // free device memory
    hipDestroyTextureObject(texObj);
    hipFreeArray(cuArray);
    hipFree(d_out);
    delete[] copiedImage;

    // Free host memory
    stbi_image_free(data);
    return 0;
}
