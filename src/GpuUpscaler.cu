#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstring>

#include "../lib/stb_image.h"
#include "../lib/stb_image_write.h"

#include "Headers.h"
#include "GpuTimer.cuh"

using namespace std;

__global__ void upscale(hipTextureObject_t originalImage, uint8_t* upscaledImage, uint32_t pixelsHandledByThread, uint32_t width, uint32_t height, uint8_t bytePerPixel, uint32_t upscaleFactor)
{
    uint32_t pixelsHandledByBlock = pixelsHandledByThread * blockDim.x;
    uint32_t startNewIndex = blockIdx.x * pixelsHandledByBlock + threadIdx.x * pixelsHandledByThread;
    uint32_t upscaledWidth = width * upscaleFactor;
    uint32_t upscaledSize = width * height * upscaleFactor * upscaleFactor;

    // iterate all pixels handled by this thread
    for (uint32_t i = 0; i < pixelsHandledByThread; i++) {
        // compute the coordinates of the pixel
        uint32_t newIndex = startNewIndex + i;

        if (newIndex < upscaledSize) {
            uint32_t x = newIndex / upscaledWidth;
            uint32_t y = newIndex - (x * upscaledWidth);

            // compute the coordinates of the pixel of the original image
            uint32_t oldX = x / upscaleFactor;
            uint32_t oldY = y / upscaleFactor;

            // copy the pixel
            uchar4  pixelToCopy = tex2D<uchar4>(originalImage, oldY, oldX);
            memcpy(&upscaledImage[newIndex * bytePerPixel], &pixelToCopy, sizeof(uchar4));
        }
    }
}

__global__ void upscaleOptimized(hipTextureObject_t originalImage, uchar4* upscaledImage, uint32_t threadsCount, uint32_t pixelsHandledByThread, uint32_t upscaledWidth, uint32_t upscaledSize, uint32_t upscaleFactor)
{
    uint32_t startNewIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // iterate all pixels handled by this thread
    for (uint32_t i = 0; i < pixelsHandledByThread; i++) {
        // compute the coordinates of the pixel
        uint32_t newIndex = startNewIndex + (i * threadsCount);

        if (newIndex < upscaledSize) {
            uint32_t x = newIndex / upscaledWidth;
            uint32_t y = newIndex - (x * upscaledWidth);

            // compute the coordinates of the pixel of the original image
            uint32_t oldX = x / upscaleFactor;
            uint32_t oldY = y / upscaleFactor;

            // copy the pixel
            uchar4 pixelToCopy = tex2D<uchar4>(originalImage, oldY, oldX);
            upscaledImage[newIndex] = pixelToCopy;
        }
    }
} 

float gpuUpscaler(size_t originalSize, size_t upscaledSize, uint8_t upscaleFactor, Settings settings, uint8_t* data, uint32_t width, uint32_t height, uint32_t bytePerPixel, string imageName)
{
    uint8_t* upscaledImage = new uint8_t[upscaledSize];

    // event based timer
    GpuTimer timer;

    // allocate GPU memory for input and output array 
    uint8_t* deviceUpscaledImage;
    hipMalloc((void**)&deviceUpscaledImage, upscaledSize);

    // create the texture object to store the original image
    hipTextureObject_t originalImage = createTextureObject(width, height, bytePerPixel, data);

    // define resources for the execution
    dim3 grid(settings.blocksPerGrid, 1, 1);                // blocks per grid
    dim3 block(settings.threadsPerBlock, 1, 1);             // threads per block

    // start kernel execution
    switch (settings.upscalerType)
    {
        case UpscalerType::UpscaleWithTextureObject:
            timer.start();
            upscale << <grid, block >> > (originalImage, deviceUpscaledImage, settings.pixelsHandledByThread, width, height, bytePerPixel, upscaleFactor);
            timer.stop();
            break;
        case UpscalerType::UpscaleWithTextureObjectOptimized:
            // precompute read-only variable
            uint32_t upscaledWidth = width * upscaleFactor;
            uint32_t upscaledSize = width * height * upscaleFactor * upscaleFactor;

            timer.start();
            upscaleOptimized << <grid, block >> > (originalImage, (uchar4*)deviceUpscaledImage, settings.threadsCount, settings.pixelsHandledByThread, upscaledWidth, upscaledSize, upscaleFactor);
            timer.stop();
            break;
    }

    // wait for the end of the execution and retrieve results from GPU memory
    hipDeviceSynchronize();
    hipMemcpy(upscaledImage, deviceUpscaledImage, upscaledSize, hipMemcpyDeviceToHost);

    // print the upscale duration
    float time = timer.getElapsedMilliseconds();
    cout << "\n---------------------------------------------------------------" << endl;
    settings.print();
    cout << "[+] (GPU) Time needed: " << time << "ms" << endl;

    // save image as PNG
    if (imageName != "") {
        cout << "[+] Saving image..." << endl;
        if (stbi_write_png(imageName.c_str(), width * upscaleFactor, height * upscaleFactor, bytePerPixel, upscaledImage, width * upscaleFactor * bytePerPixel))
            cout << "[+] Image saved successfully" << endl;
        else
            cout << "[-] Failed to save image" << endl;
    }

    // free memory
    delete[] upscaledImage;
    hipFree(deviceUpscaledImage);

    return time;
}
