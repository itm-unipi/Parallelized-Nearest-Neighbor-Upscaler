#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <cstring>

#include "../lib/stb_image.h"
#include "../lib/stb_image_write.h"

#include "Headers.h"
#include "GpuTimer.cuh"

using namespace std;

__global__ void upscaleFromOriginalImage(uint8_t* imageToUpscale, uint8_t* upscaledImage, uint32_t width, uint8_t upscaleFactor, uint8_t bytePerPixel)
{
    // get the pixel position in the original image vector
    uint32_t oldIndex = ((((blockIdx.y * gridDim.x + blockIdx.x) * blockDim.y + threadIdx.y) * blockDim.x) + threadIdx.x) * bytePerPixel;

    // convert the position in a matrix notation
    uint32_t i = oldIndex / (width * bytePerPixel);
    uint32_t j = oldIndex - (i * width * bytePerPixel);

    // compute the position of the first pixel to duplicate in upscaled image
    uint32_t newi = i * upscaleFactor;
    uint32_t newj = j * upscaleFactor;
    uint32_t upscaledWidth = width * upscaleFactor;

    // iterate the pixel to duplicate in upscaled image
    for (int m = newi; m < newi + upscaleFactor; m++) {
        for (int n = newj; n < newj + upscaleFactor * bytePerPixel; n += bytePerPixel) {
            // compute the pixel position in the upscaled image vector
            uint32_t newIndex = m * upscaledWidth * bytePerPixel + n;
            
            // manage single channel if tridimensional version, else manage all the others
            if (blockDim.z == 1) {
                for (int k = 0; k < bytePerPixel; k++)
                    upscaledImage[newIndex + k] = imageToUpscale[oldIndex + k];
            } else {
                upscaledImage[newIndex + threadIdx.z] = imageToUpscale[oldIndex + threadIdx.z];
            }
        }
    }
}

__global__ void upscaleFromUpscaledImage(uint8_t* imageToUpscale, uint8_t* upscaledImage, uint32_t width, uint8_t upscaleFactor, uint8_t bytePerPixel)
{
    // get the pixel position in the upscaled image vector
    uint32_t newIndex = ((((blockIdx.y * gridDim.x + blockIdx.x) * blockDim.y + threadIdx.y) * blockDim.x) + threadIdx.x) * bytePerPixel;

    // convert the position in a matrix notation
    uint32_t newi = newIndex / (width * upscaleFactor * bytePerPixel);
    uint32_t newj = (newIndex - (newi * width * upscaleFactor * bytePerPixel)) / bytePerPixel;

    // compute the position of the pixel to copy from the original image
    uint32_t i = newi / upscaleFactor;
    uint32_t j = newj / upscaleFactor;
    uint32_t oldIndex = (i * width + j) * bytePerPixel;

    // manage single channel if tridimensional version, else manage all the others
    if (blockDim.z == 1) {
        for (int k = 0; k < bytePerPixel; k++)
            upscaledImage[newIndex + k] = imageToUpscale[oldIndex + k];
    } else {
        upscaledImage[newIndex + threadIdx.z] = imageToUpscale[oldIndex + threadIdx.z];
    }        
}

void gpuUpscaler(size_t originalSize, size_t upscaledSize, uint8_t upscaleFactor, Settings settings, uint8_t* data, uint32_t width, uint32_t height, uint32_t bytePerPixel, string imageName)
{
    uint8_t* upscaledImage = new uint8_t[upscaledSize];

    // event based timer
    GpuTimer timer;

    // allocate GPU memory for input and output array 
    uint8_t* d_data, * d_out;
    hipMalloc((void**)&d_data, originalSize);
    hipMalloc((void**)&d_out, upscaledSize);
    hipMemcpy(d_data, data, originalSize, hipMemcpyHostToDevice);

    // define resources for the execution
    dim3 grid(settings.blocksPerGridX, settings.blocksPerGridY, settings.blocksPerGridZ);               // blocks per grid
    dim3 block(settings.threadsPerBlockX, settings.threadsPerBlockY, settings.threadsPerBlockZ);        // threads per block

    // start kernel execution
    timer.start();
    switch (settings.upscalerType)
    {
        case UpscalerType::UpscaleFromOriginalImage:
            upscaleFromOriginalImage << <grid, block >> > (d_data, d_out, width, upscaleFactor, bytePerPixel);
            break;
        case UpscalerType::UpscaleFromUpscaledImage:
            upscaleFromUpscaledImage << <grid, block >> > (d_data, d_out, width, upscaleFactor, bytePerPixel);
            break;
    }
    timer.stop();

    // wait for the end of the execution and retrieve results from GPU memory
    hipDeviceSynchronize();
    hipMemcpy(upscaledImage, d_out, upscaledSize, hipMemcpyDeviceToHost);

    // print the upscale duration
    float time = timer.getElapsedMilliseconds();
    cout << "\n---------------------------------------------------------------" << endl;
    settings.print();
    cout << "[+] (GPU) Time needed: " << time << "ms" << endl;

    // save image as PNG
    if (imageName != "") {
        cout << "[+] Saving image..." << endl;
        if (stbi_write_png(imageName.c_str(), width * upscaleFactor, height * upscaleFactor, bytePerPixel, upscaledImage, width * upscaleFactor * bytePerPixel))
            cout << "[+] Image saved successfully" << endl;
        else
            cout << "[-] Failed to save image" << endl;
    }

    // free memory
    delete[] upscaledImage;
    hipFree(d_data);
    hipFree(d_out);
}
